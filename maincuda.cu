#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

using namespace cv;


__global__ void blurEffect(int *R, int *G, int *B, int *RED, int *GREEN, int *BLUE, int size, int rows, int cols, int kernel, int NTHREADS)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  //using just for "for loop"
  //int start = id*size/NTHREADS;
  //int end = (id+1)*size/NTHREADS;

  int tmp_red = 0;
  int tmp_green = 0;
  int tmp_blue = 0;

  /*
  for (int i = start; i < end; ++i)
  {
    tmp_red = 0;
    tmp_green = 0;
    tmp_blue = 0;

    if ( (i-cols)>0 ) {
      tmp_red += R[i-cols*kernel];
      tmp_green += G[i-cols*kernel];
      tmp_blue += B[i-cols*kernel];
    }

    if ( (i+cols)<size ) {
      tmp_red += R[i+cols*kernel];
      tmp_green += G[i+cols*kernel];
      tmp_blue += B[i+cols*kernel];
    }

    if ( i%cols == 0 ) {
      tmp_red += R[i+kernel];
      tmp_green += G[i+kernel];
      tmp_blue += B[i+kernel];
    }
    else if ( i%cols == cols-1 ) {
      tmp_red += R[i-kernel];
      tmp_green += G[i-kernel];
      tmp_blue += B[i-kernel];
    }
    else {
      tmp_red += R[i+kernel] + R[i-kernel];
      tmp_green += G[i+kernel] + G[i-kernel];
      tmp_blue += B[i+kernel]+ B[i-kernel];
    }

    RED[i] = tmp_red/4;
    GREEN[i] = tmp_green/4;
    BLUE[i] = tmp_blue/4;
  }*/

  if (id < size)
  {
    tmp_red = 0;
    tmp_green = 0;
    tmp_blue = 0;

    if ( (id-cols*kernel) > 0 ) {
      tmp_red += R[id-cols*kernel];
      tmp_green += G[id-cols*kernel];
      tmp_blue += B[id-cols*kernel];
    }

    if ( (id+cols*kernel) < size ) {
      tmp_red += R[id+cols*kernel];
      tmp_green += G[id+cols*kernel];
      tmp_blue += B[id+cols*kernel];
    }

    if ( id%cols == 0 ) {
      tmp_red += R[id+kernel];
      tmp_green += G[id+kernel];
      tmp_blue += B[id+kernel];
    }
    else if ( id%cols == cols-1 ) {
      tmp_red += R[id-kernel];
      tmp_green += G[id-kernel];
      tmp_blue += B[id-kernel];
    }
    else {
      tmp_red += R[id+kernel] + R[id-kernel];
      tmp_green += G[id+kernel] + G[id-kernel];
      tmp_blue += B[id+kernel]+ B[id-kernel];
    }

    RED[id] = tmp_red/4;
    GREEN[id] = tmp_green/4;
    BLUE[id] = tmp_blue/4;
  }

}

void randomFill(int *V, int row, int col){
  for(int i=0 ; i<row ; i++){
    for(int j=0 ; j<col ; j++){
      V[i*col+j] = 0;
    }
  }
}

void checkError(string s,hipError_t err){
	if(err != hipSuccess){
		std::cout<<s<<" "<<hipGetErrorString(err)<<std::endl;
		exit(EXIT_FAILURE);
	}
}

int main( int argc, char** argv )
{
  char* imageName = argv[1];
  Mat image;
  image = imread( imageName, IMREAD_COLOR );
  if( !image.data )
  {
    std::cout<<" No image data \n ";
    return -1;
  }

  Mat blur = image.clone();

  int radius;
  size_t the_size = image.rows * image.cols * sizeof(int);
  int total_size = image.rows * image.cols;

  //TODO using for "for loop" inside __global__ function
  //int BLOCKS = 2;
  int NTHREADS = 192;
  int threadsPerBlock = 256;

  if(argv[2]==NULL || argc<3) {
    radius = 3;
    NTHREADS = 192;
    threadsPerBlock = 256;
    std::cout<<"Radius and CUDA NUM_THREADS are null"<<std::endl;
  }
  else {
    radius = atoi(argv[2]);
  }

	if (argv[3]==NULL || argc<4) {
		NTHREADS = 192;
    threadsPerBlock = 256;
		std::cout<<"CUDA NUM_THREADS is null"<<std::endl;
	}
	else {
		NTHREADS = atoi(argv[3]);
    threadsPerBlock = atoi(argv[3]);
	}

  int blocksPerGrid =(total_size + threadsPerBlock - 1) / threadsPerBlock;

  //take the time
  struct timespec start, finish;
  double elapsed;

  //error variable for cuda
  hipError_t err = hipSuccess;

  int *h_R, *h_G, *h_B, *h_RED, *h_GREEN, *h_BLUE;
  int *d_R, *d_G, *d_B, *d_RED, *d_GREEN, *d_BLUE;

  //Memory in Host
  h_R = new int[the_size];
  h_G = new int[the_size];
  h_B = new int[the_size];
  h_RED = new int[the_size];
  h_GREEN = new int[the_size];
  h_BLUE = new int[the_size];

  //Fill Arrays RED GREEN BLUE with zero
  randomFill(h_RED, image.rows, image.cols);
  randomFill(h_GREEN, image.rows, image.cols);
  randomFill(h_BLUE, image.rows, image.cols);

  //Fill arrays RGB with data image
  //TODO
  for (int i = 0; i < image.rows; ++i)
  {
    for (int j = 0; j < image.cols; ++j)
    {
      h_R[i*image.cols+j] = image.at<Vec3b>(i,j)[0];
      h_G[i*image.cols+j] = image.at<Vec3b>(i,j)[1];
      h_B[i*image.cols+j] = image.at<Vec3b>(i,j)[2];
    }
  }

  //Memory for cuda in video device
  err = hipMalloc((void**)&d_R, the_size);
  checkError("Error al reservar memoria para R",err);

  err = hipMalloc((void**)&d_G, the_size);
  checkError("Error al reservar memoria para G",err);

  err = hipMalloc((void**)&d_B, the_size);
  checkError("Error al reservar memoria para B",err);

  err = hipMalloc((void**)&d_RED, the_size);
  checkError("Error al reservar memoria para RED",err);

  err = hipMalloc((void**)&d_GREEN, the_size);
  checkError("Error al reservar memoria para GREEN",err);

  err = hipMalloc((void**)&d_BLUE, the_size);
  checkError("Error al reservar memoria para BLUE",err);

  //Copy data matrixes RGB from Host to video device

  err = hipMemcpy(d_R, h_R, the_size, hipMemcpyHostToDevice);
  checkError("Error al pasar los datos de RED al device", err);

  err = hipMemcpy(d_G, h_G, the_size, hipMemcpyHostToDevice);
  checkError("Error al pasar los datos de GREEN al device", err);

  err = hipMemcpy(d_B, h_B, the_size, hipMemcpyHostToDevice);
  checkError("Error al pasar los datos de BLUE al device", err);

  //take the time since execute function
  clock_gettime(CLOCK_MONOTONIC, &start);

  //New Test with more capacity and using if id in __global__ function
  //integer variables declared above
  //int threadsPerBlock = 256;
  //int blocksPerGrid =(total_size + threadsPerBlock - 1) / threadsPerBlock;
  blurEffect<<<blocksPerGrid, threadsPerBlock>>>(d_R,d_G,d_B,d_RED,d_GREEN,d_BLUE,total_size,image.rows,image.cols, radius, NTHREADS);

  //Execute cuda function using for loop
  //blurEffect<<<BLOCKS,NTHREADS/BLOCKS>>>(d_R,d_G,d_B,d_RED,d_GREEN,d_BLUE,total_size,image.rows,image.cols, radius, NTHREADS);

  err = hipGetLastError();
  checkError("Error al ejecutar el kernel",err);

  //Cuda memory to host
  err = hipMemcpy(h_RED, d_RED, the_size, hipMemcpyDeviceToHost);
  checkError("Error al pasar los datos de R al device", err);

  err = hipMemcpy(h_GREEN, d_GREEN, the_size, hipMemcpyDeviceToHost);
  checkError("Error al pasar los datos de G al device", err);

  err = hipMemcpy(h_BLUE, d_BLUE, the_size, hipMemcpyDeviceToHost);
  checkError("Error al pasar los datos de B al device", err);

  //Move memory to new Mat vector
  for (int i = 0; i < image.rows; ++i)
	{
		for (int j = 0; j < image.cols; ++j)
		{
			blur.at<Vec3b>(i,j)[0] = h_RED[i*image.cols+j];
			blur.at<Vec3b>(i,j)[1] = h_GREEN[i*image.cols+j];
			blur.at<Vec3b>(i,j)[2] = h_BLUE[i*image.cols+j];
		}
	}

  //take the time
  clock_gettime(CLOCK_MONOTONIC, &finish);
  elapsed = (finish.tv_sec - start.tv_sec);
  elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
  std::cout<<"Time: "<< elapsed << std::endl;

  //write in file
  std::ofstream output;
  output.open("maincudaResult.txt", std::ofstream::app | std::ofstream::out );
  output<< imageName <<", "<< radius << ", " << threadsPerBlock <<", "<< elapsed << std::endl;
  output.close();

  //free Memory in device
	err=hipFree(d_R);
	checkError("Error al liberar la memoria del device R",err);
	err=hipFree(d_G);
	checkError("Error al liberar la memoria del device G",err);
	err=hipFree(d_B);
	checkError("Error al liberar la memoria del device B",err);
	err=hipFree(d_RED);
	checkError("Error al liberar la memoria del device RED",err);
  err=hipFree(d_GREEN);
	checkError("Error al liberar la memoria del device GREEN",err);
  err=hipFree(d_BLUE);
	checkError("Error al liberar la memoria del device BLUE",err);

	err = hipDeviceReset();
	checkError("Error al resetear el device",err);

  //free memory in host
  delete h_R;
  delete h_G;
  delete h_B;
  delete h_RED;
  delete h_GREEN;
  delete h_BLUE;

  //write the blur image
  imwrite( "blurimage.jpg", blur );
  //namedWindow( "Blur image", WINDOW_AUTOSIZE );
  //imshow( "Blur image", blur );
  waitKey(0);

  return 0;
}
